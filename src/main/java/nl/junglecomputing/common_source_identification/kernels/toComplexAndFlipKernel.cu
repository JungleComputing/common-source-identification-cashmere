
#include <hip/hip_runtime.h>
// fermi

/*
 * Copyright 2018 Vrije Universiteit Amsterdam, The Netherlands
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0

 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

extern "C" {
    __global__ void toComplexAndFlipKernel(const int h, const int w, float* output, const float* input);
}


__global__ void toComplexAndFlipKernel(const int h, const int w, float* output, const float* input) {
    const int i = blockIdx.y;
    const int bj = blockIdx.x;
    const int wtj = threadIdx.y;
    const int ttj = threadIdx.x;

    const int nrThreadsW = min(1024, w);
    const int nrThreadsNrThreadsW = min(32, nrThreadsW);
    const int tj = wtj * (1 * nrThreadsNrThreadsW) + ttj;
    if (tj < nrThreadsW) {
        const int j = bj * (1 * nrThreadsW) + tj;
        if (j < w) {
            const int oi = h - i - 1;
            const int oj = w - j - 1;
            output[(oj + oi * (1 * w)) * 2 + 0] = input[j + i * (1 * w)];
            output[(oj + oi * (1 * w)) * 2 + 1] = 0.0;
        }
    }
}
