
#include <hip/hip_runtime.h>
// fermi

/*
 * Copyright 2018 Vrije Universiteit Amsterdam, The Netherlands
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0

 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

extern "C" {
    __global__ void varianceZeroMeanKernel(const int n, float* variance, const float* input);
}


__global__ void varianceZeroMeanKernel(const int n, float* variance, const float* input) {
    const int wti = threadIdx.y;
    const int tti = threadIdx.x;

    const int nrThreads = 1024;
    const int nrThreadsNrThreads = min(32, nrThreads);
    __shared__ float reduceMem[1024];
    const int ti = wti * (1 * nrThreadsNrThreads) + tti;
    if (ti < nrThreads) {
        if (ti < n) {
            float sum = 0.0;
            for (int i = ti; i < n; i += nrThreads) {
                sum += input[i] * input[i];
            }
            reduceMem[ti] = sum;
            __syncthreads();
            for (int i = nrThreads / 2; i > 0; i >>= 1) {
                if (ti < i) {
                    reduceMem[ti] += reduceMem[ti + i];
                }
                __syncthreads();
            }
            if (ti == 0) {
                *variance = reduceMem[0] * n / (n - 1);
            }
        }
    }
}
