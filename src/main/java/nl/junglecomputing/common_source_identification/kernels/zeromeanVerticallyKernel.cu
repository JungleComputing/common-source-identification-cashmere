
#include <hip/hip_runtime.h>
// fermi

/*
 * Copyright 2018 Vrije Universiteit Amsterdam, The Netherlands
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0

 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


extern "C" {
    __global__ void zeromeanVerticallyKernel(const int h, const int w, float* output, const float* input);
}

__global__ void zeromeanVerticallyKernel(const int h, const int w, float* output, const float* input) {
    const int bj = blockIdx.x;
    const int wtj = threadIdx.y;
    const int ttj = threadIdx.x;

    const int nrThreadsW = min(1024, w);
    const int nrThreadsNrThreadsW = min(32, nrThreadsW);
    const int tj = wtj * (1 * nrThreadsNrThreadsW) + ttj;
    if (tj < nrThreadsW) {
        const int j = bj * (1 * nrThreadsW) + tj;
        if (j < w) {
            float sumEven = 0.0;
            float sumOdd = 0.0;
            for (int i = 0; i < h - 1; i += 2) {
                sumEven += input[j + i * (1 * w)];
                sumOdd += input[j + (i + 1) * (1 * w)];
            }
            const float meanEven = sumEven / ((h + 1) / 2);
            const float meanOdd = sumOdd / (h / 2);
            for (int i = 0; i < h - 1; i += 2) {
                output[j + i * (1 * w)] = input[j + i * (1 * w)] - meanEven;
                output[j + (i + 1) * (1 * w)] = input[j + (i + 1) * (1 * w)] - meanOdd;
            }
        }
    }
}
