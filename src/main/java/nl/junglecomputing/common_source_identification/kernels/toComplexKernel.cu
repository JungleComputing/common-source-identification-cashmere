
#include <hip/hip_runtime.h>
// fermi

/*
 * Copyright 2018 Vrije Universiteit Amsterdam, The Netherlands
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0

 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

extern "C" {
    __global__ void toComplexKernel(const int n, float* output, const float* input);
}

__global__ void toComplexKernel(const int n, float* output, const float* input) {
    const int bi = blockIdx.x;;
    const int wti = threadIdx.y;
    const int tti = threadIdx.x;

    const int nrThreadsN = min(1024, n);
    const int nrThreadsNrThreadsN = min(32, nrThreadsN);
    const int ti = wti * (1 * nrThreadsNrThreadsN) + tti;
    if (ti < nrThreadsN) {
        const int i = bi * (1 * nrThreadsN) + ti;
        if (i < n) {
            output[i * 2 + 0] = input[i];
            output[i * 2 + 1] = 0.0;
        }
    }
}
